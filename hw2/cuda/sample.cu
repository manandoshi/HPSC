
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void add(int* d_a, int* d_b, int* d_c) {
    int t = threadIdx.x;
    int index = t + blockIdx.x*blockDim.x;
    d_c[index] = d_a[index] + d_b[index];
}


int main( void) {
    const int N = 512;
    const int M = 64;
    int size = N*sizeof(int);
    int *a, *b, *c;
    int *d_a, *d_b, *d_c;

    a = (int*)malloc(size);
    b = (int*)malloc(size);
    c = (int*)malloc(size);

    for(int i = 0; i<N; i++){
        a[i] = i;
        b[i] = 2*i;
    }

    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    add<<<N/M,M>>>(d_a, d_b, d_c);

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    for(int i = 0; i<N; i++){
        printf ("%d \t",c[i]);
    }
    printf("\n");

    return 0;
}
