
#include <hip/hip_runtime.h>
#include<stdlib.h>
#include<stdio.h>
#include<math.h>
#include<time.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define TPB 512

/** Need to use this function instead of atomicAdd for double precision. Computation done in single precision(float) for now**/
__device__ double atomicAdd_double(double* address, double val)
{
    unsigned long long int* address_as_ull =
                             (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                               __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}


__global__ void trap(float *out, int N) {
    int index = threadIdx.x + blockIdx.x*blockDim.x;
    if(index>=N){
        return;
    }
    __shared__ float temp[TPB];

    temp[threadIdx.x] = sin(M_PI*(float)index/((float)N-1));

    __syncthreads();
    if(threadIdx.x == 0){
        float sum = 0;
        for(int i = index; i<min(N,TPB); i++){
            sum += temp[i-index];
        }
        atomicAdd(out, sum);
    }
    return;
}


__global__ void markov(float *out, int N) {

    int index = threadIdx.x + blockIdx.x*blockDim.x;

    if(index>=N){
        return;
    }

    hiprandState_t state;
    hiprand_init(0,
                index,
                0,
                &state);

    __shared__ float temp[TPB];
    temp[index] = sin(hiprand_uniform(&state)*M_PI);

    __syncthreads();

    if(threadIdx.x == 0){
        float sum = 0;
        for(int i = index; i< min(N,index+TPB); i++){
            sum += temp[i]/N;
        }
        atomicAdd(out, sum);
    }
    return;
}

int main(int argc, char** argv){

    int num_points = atoi(argv[2]);
    int num_blocks = (num_points+TPB-1)/TPB;

    if(argv[1][0]=='T'){
        clock_t begin = clock();

        float integral = 1;
        float* d_integral;

        hipMalloc((void**)&d_integral, sizeof(float));
        trap<<<num_blocks,TPB>>>(d_integral, num_points);

        hipMemcpy(d_integral, &integral, sizeof(float), hipMemcpyDeviceToHost);

        clock_t end = clock();
        double time_spent = (double)(end-begin)/CLOCKS_PER_SEC;
        float error = integral > 2.0 ? integral - 2.0 : 2.0 - integral;

        printf("Trapezoidal, %.9f, %.9f,%d, %d,%.9f\n",error, integral, num_blocks, num_points, time_spent);
    }
    /*
    else if(argv[1][0]=='M'){
        srand(time(NULL) + world_rank);
        clock_t begin = clock();
        double integral = 0;
        for(int i=np*world_rank;i<np*(world_rank+1); i++){
            double x = rand()*M_PI/(double)RAND_MAX;
            integral += sin(x);
        }
        integral = integral*M_PI/num_points;
        //printf("Processor %d reporting: integral = %f \n",world_rank,integral);

        MPI_Status status;
        if(world_rank == 0){
            for(int i=1; i<world_size;i++){
                double other_integral;
                MPI_Recv( &other_integral, 1, MPI_DOUBLE, i, MPI_ANY_TAG, MPI_COMM_WORLD, &status);
                integral+= other_integral;
            }
            clock_t end = clock();
            double time_spent = (double)(end-begin)/CLOCKS_PER_SEC;
            double error = integral > 2.0 ? integral - 2.0 : 2.0 - integral;
            printf("Monte-Carlo, %.9f, %.9f,%d, %d,%.9f\n",error, integral, world_size, num_points, time_spent);
        }
        else{
            MPI_Send( &integral, 1, MPI_DOUBLE, 0,world_rank, MPI_COMM_WORLD);
        }

        MPI_Finalize();
    }
    */
    return 0;
}
