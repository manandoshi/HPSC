
#include <hip/hip_runtime.h>
#include<stdlib.h>
#include<stdio.h>
#include<math.h>
#include<time.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define TPB 512

/** Need to use this function instead of atomicAdd for double precision. Computation done in single precision(float) for now**/
__device__ double atomicAdd_double(double* address, double val)
{
    unsigned long long int* address_as_ull =
                             (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                               __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}

__global__ void trap(float *out, int N) {

    __shared__ float temp[TPB];

    temp[threadIdx.x] = 0;

    int index = threadIdx.x + blockIdx.x*blockDim.x;

    if(index>=N){
    }
    else{
        temp[threadIdx.x] = sin(M_PI*(float)index/((float)N-1));
    }

    __syncthreads();

    if(threadIdx.x == 0){
        float sum = 0;

        for(int i = 0; i<TPB; i++){
            sum += temp[i];
        }

        sum = sum*M_PI/(N-1);
        atomicAdd(out, sum);
    }
    return;
}

__global__ void mc(float *out, int N, int seed) {

    __shared__ float temp[TPB];

    temp[threadIdx.x] = 0;

    int index = threadIdx.x + blockIdx.x*blockDim.x;

    if(index>=N){
    }
    else{
        hiprandState_t state;
        hiprand_init(blockIdx.x + seed,
                    threadIdx.x,
                    0,
                    &state);

        temp[threadIdx.x] = sin(hiprand_uniform(&state)*M_PI);
    }

    __syncthreads();

    if(threadIdx.x == 0){
        float sum = 0;
        for(int i = 0; i<TPB; i++){
            sum += temp[i];
        }
        sum = sum*M_PI/N;
        atomicAdd(out, sum);
    }
    return;
}

int main(int argc, char** argv){

    int num_points = atoi(argv[2]);
    int num_blocks = (num_points+TPB-1)/TPB;

    if(argv[1][0]=='T'){
        clock_t begin = clock();

        float* integral;
        float* d_integral;

        hipMalloc((void**)&d_integral, sizeof(float));
        integral = (float *)malloc(sizeof(float));

        trap<<<num_blocks,TPB>>>(d_integral, num_points);

        hipMemcpy(integral, d_integral, sizeof(float), hipMemcpyDeviceToHost);
        
        clock_t end = clock();
        double time_spent = (double)(end-begin)/CLOCKS_PER_SEC;
        float error = *integral > 2.0 ? *integral - 2.0 : 2.0 - *integral;
        
        printf("Trapezoidal, %.12f, %.12f,%d, %d,%.12f\n",error, *integral, num_blocks, num_points, time_spent);
        
        free(integral);
        hipFree(d_integral);
    }

    else if(argv[1][0]=='M'){
        clock_t begin = clock();

        float* integral;
        float* d_integral;

        hipMalloc((void**)&d_integral, sizeof(float));
        integral = (float *)malloc(sizeof(float));

        mc<<<num_blocks,TPB>>>(d_integral, num_points, time(NULL));

        hipMemcpy(integral, d_integral, sizeof(float), hipMemcpyDeviceToHost);
        
        clock_t end = clock();
        double time_spent = (double)(end-begin)/CLOCKS_PER_SEC;
        float error = *integral > 2.0 ? *integral - 2.0 : 2.0 - *integral;
        
        printf("Monte-Carlo, %.12f, %.12f,%d, %d,%.12f\n",error, *integral, num_blocks, num_points, time_spent);
        
        free(integral);
        hipFree(d_integral);
    }
    return 0;
}
