
#include <hip/hip_runtime.h>
#include<stdlib.h>
#include<stdio.h>
#include<math.h>
#include<time.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define TPB 512

__device__ double atomicAdd_double(double* address, double val)
{
    unsigned long long int* address_as_ull =
                             (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                               __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}


__global__ void trap(double *out, int *N) {
    int index = threadIdx.x + blockIdx.x*blockDim.x;
    if(index>=*N){
        return;
    }
    __shared__ double temp[TPB];

    temp[index] = sin(M_PI*(double)index/((double)*N-1));

    __syncthreads();
    if(threadIdx.x == 0){
        double sum = 0;
        for(int i = index; i< min(*N,index+TPB); i++){
            sum += temp[i];
        }
        atomicAdd_double(out, sum);
    }
    return;
}



int main(int argc, char** argv){

    int num_points = atoi(argv[2]);

    int num_blocks = num_points/TPB + 1;


    if(argv[1][0]=='T'){
        clock_t begin = clock();

        double integral;
        double* d_integral;
        int* d_N;

        hipMalloc((void**)&d_N, sizeof(int));
        hipMalloc((void**)&d_integral, sizeof(double));
        
        hipMemcpy(d_N, &num_points, sizeof(int), hipMemcpyHostToDevice);

        trap<<<num_blocks,TPB>>>(d_integral, d_N);

        hipMemcpy(&integral, d_integral, sizeof(double), hipMemcpyDeviceToHost);

        clock_t end = clock();
        double time_spent = (double)(end-begin)/CLOCKS_PER_SEC;
        double error = integral > 2.0 ? integral - 2.0 : 2.0 - integral;

        printf("Trapezoidal, %.9f, %.9f,%d, %d,%.9f\n",error, integral, num_blocks, num_points, time_spent);
    }
    /*
    else if(argv[1][0]=='M'){
        srand(time(NULL) + world_rank);
        clock_t begin = clock();
        double integral = 0;
        for(int i=np*world_rank;i<np*(world_rank+1); i++){
            double x = rand()*M_PI/(double)RAND_MAX;
            integral += sin(x);
        }
        integral = integral*M_PI/num_points;
        //printf("Processor %d reporting: integral = %f \n",world_rank,integral);

        MPI_Status status;
        if(world_rank == 0){
            for(int i=1; i<world_size;i++){
                double other_integral;
                MPI_Recv( &other_integral, 1, MPI_DOUBLE, i, MPI_ANY_TAG, MPI_COMM_WORLD, &status);
                integral+= other_integral;
            }
            clock_t end = clock();
            double time_spent = (double)(end-begin)/CLOCKS_PER_SEC;
            double error = integral > 2.0 ? integral - 2.0 : 2.0 - integral;
            printf("Monte-Carlo, %.9f, %.9f,%d, %d,%.9f\n",error, integral, world_size, num_points, time_spent);
        }
        else{
            MPI_Send( &integral, 1, MPI_DOUBLE, 0,world_rank, MPI_COMM_WORLD);
        }

        MPI_Finalize();
    }
    */
    return 0;
}
